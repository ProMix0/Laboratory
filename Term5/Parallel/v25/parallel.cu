
#include <hip/hip_runtime.h>
#include <algorithm>
#include <fstream>
#include <iostream>
#include <limits.h>
#include <stdio.h>
#include <sys/time.h>
#include <vector>

using namespace std;

#define THREADS_PER_BLOCK 64
#define GRAVITATIONAL_CONSTANT 6.7e-11
#define TIME_STEP 0.001f
#define PI 3.14152926f
#define RADIUS 10

struct Body {
  float3 pos;
  float3 vel;
  float3 acc;
  float m;
};

__device__ void updateBody(Body &body) {
  body.vel.x += body.acc.x * TIME_STEP;
  body.vel.y += body.acc.y * TIME_STEP;
  body.vel.z += body.acc.z * TIME_STEP;

  body.pos.x += body.vel.x * TIME_STEP;
  body.pos.y += body.vel.y * TIME_STEP;
  body.pos.z += body.vel.z * TIME_STEP;
}

__device__ void calculateBodyNewState(Body *bodies, int count, int id) {
  bodies[id].acc.x = 0;
  bodies[id].acc.y = 0;
  bodies[id].acc.z = 0;

  for (int i = 0; i < count; i++) {
    if (i == id)
      continue;

    float3 distance;
    distance.x = bodies[i].pos.x - bodies[id].pos.x;
    distance.y = bodies[i].pos.y - bodies[id].pos.y;
    distance.z = bodies[i].pos.z - bodies[id].pos.z;

    float dist_sqr = distance.x * distance.x + distance.y * distance.y +
                     distance.z * distance.z;

    bodies[id].acc.x += bodies[i].m / dist_sqr;
    bodies[id].acc.y += bodies[i].m / dist_sqr;
    bodies[id].acc.z += bodies[i].m / dist_sqr;
  }

  bodies[id].acc.x *= GRAVITATIONAL_CONSTANT;
  bodies[id].acc.y *= GRAVITATIONAL_CONSTANT;
  bodies[id].acc.z *= GRAVITATIONAL_CONSTANT;
}

__global__ void nbody(Body *bodies, int count) {
  int id = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
  if (id >= count)
    return;

  calculateBodyNewState(bodies, count, id);
}

__global__ void updateBodies(Body *bodies, int count) {
  int id = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
  if (id >= count)
    return;

  updateBody(bodies[id]);
}

void getNumberOfBlocksAndThreads(int elemsCount, int *blocks, int *threads) {
  *blocks = ((elemsCount - 1) / THREADS_PER_BLOCK) ?: 1;
  *threads = (elemsCount < THREADS_PER_BLOCK ? elemsCount : THREADS_PER_BLOCK);
}

void createBodies(vector<Body> &bodies, int count) {
  bodies = vector<Body>(count);
  if (getenv("SEED"))
    srand(atoi(getenv("SEED")));
  else
    srand((0));

  for (int i = 0; i < bodies.size(); i++) {
    bodies[i].pos.x = rand();
    bodies[i].pos.y = rand();
    bodies[i].pos.z = rand();

    bodies[i].m = rand() / ((float)RAND_MAX / 100000);

    bodies[i].vel.x = 0;
    bodies[i].vel.y = 0;
    bodies[i].vel.z = 0;

    bodies[i].acc.x = 0;
    bodies[i].acc.y = 0;
    bodies[i].acc.z = 0;
  }
}

void writeResultsToFile(vector<Body> &bodies) {
  std::ofstream fout("result_parallel");
  if (!fout) {
    printf("fail to open result_parallel\n");
    return;
  }

  fout << "Bodies: \n";

  for (int i = 0; i < bodies.size(); i++) {
    fout << "(x:" << bodies[i].pos.x << "| y:" << bodies[i].pos.y
         << "| z:" << bodies[i].pos.z << ")\n";
  }
}

int main(int argc, char **argv) {
  if (argc < 3) {
    printf("Usage: ./parallel <N> <time>\n");
    return -1;
  }

  int count = atoi(argv[1]);
  if (count < 2) {
    printf("Must be at least two bodies\n");
    return -1;
  }

  int time = atoi(argv[2]);
  if (time < TIME_STEP) {
    printf("Too small time\n");
    return -1;
  }

  vector<Body> bodies;
  createBodies(bodies, count);

  struct timeval start, end;

  Body *d_bodies;
  hipMalloc(&d_bodies, bodies.size() * sizeof(Body));
  hipMemcpy(d_bodies, &bodies.front(), bodies.size() * sizeof(Body),
             hipMemcpyHostToDevice);

  float curTime = 0;

  int blocks, threads;
  getNumberOfBlocksAndThreads(bodies.size(), &blocks, &threads);

  gettimeofday(&start, 0);
  while (curTime < time) {
    nbody<<<blocks, threads>>>(d_bodies, bodies.size());
    updateBodies<<<blocks, threads>>>(d_bodies, bodies.size());

    curTime += TIME_STEP;
  }
  hipDeviceSynchronize();
  gettimeofday(&end, 0);

  hipMemcpy(&bodies.front(), d_bodies, bodies.size() * sizeof(Body),
             hipMemcpyDeviceToHost);
  hipFree(d_bodies);

  double time_result =
      (end.tv_sec - start.tv_sec) + (end.tv_usec - start.tv_usec) / 1e6;
  printf("Time: %lf\n", time_result);

  writeResultsToFile(bodies);
  return 0;
}
